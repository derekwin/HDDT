#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <nn_kernel.cuh>
#include <stdio.h>

__global__ void add(int n, float *arrayX, float *arrayY) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n)
    arrayY[index] = arrayX[index] + arrayY[index];
}

__global__ void initArrayKernel(int n, float *arr, float value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    arr[idx] = value;
  }
}

void KernelAdd(int n, float *arrayX, float *arrayY, int blockSize) {
  int numBlocks = (n + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(n, arrayX, arrayY);
  hipDeviceSynchronize();
}

void KernelInit(int n, float *arr, int blockSize, float value) {
  int numBlocks = (n + blockSize - 1) / blockSize;
  hipMalloc((void **)&arr, n * sizeof(float));
  initArrayKernel<<<numBlocks, blockSize>>>(n, arr, value);
  hipDeviceSynchronize();
}

void KernelMemcpy(int n, float *arrayS, float *arrayD) {
  hipMemcpy(arrayD, arrayS, n * sizeof(float), hipMemcpyDeviceToHost);
}

void KernleInitDriver(int index) {
  hipError_t cudaStatus = hipSetDevice(index);
  if (cudaStatus != hipSuccess) {
    printf("Error: hipSetDevice failed: %s", hipGetErrorString(cudaStatus));
    return;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  printf("Using device: %s\n", deviceProp.name);
}
